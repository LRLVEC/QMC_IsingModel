#include "hip/hip_runtime.h"
#include <_Time.h>
#include <random>
#include <hip/hip_runtime.h>
#include <>
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#include <hiprand/hiprand_kernel.h>

template<class T, unsigned long long blockSize>__device__ void warpReduce(volatile T* sdata, unsigned int tid)
{
	if constexpr (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if constexpr (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if constexpr (blockSize >= 16) sdata[tid] += sdata[tid + 8];
	if constexpr (blockSize >= 8) sdata[tid] += sdata[tid + 4];
	if constexpr (blockSize >= 4) sdata[tid] += sdata[tid + 2];
	if constexpr (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}
template<class T, unsigned long long blockSize>__global__ void reduction(T* a, T* b, unsigned int N)//N % (2*blockSize) == 0
{
	unsigned int const tid(threadIdx.x);
	unsigned int i(blockIdx.x * blockSize * 2 + tid);
	unsigned int const gridSize(2 * blockSize * gridDim.x);
	T ans(0);
	while (i < N)
	{
		//if (i + blockSize < N)
		ans += a[i] + a[i + blockSize];
		//else
		//	ans += a[i];
		i += gridSize;
	}
	__shared__ T sdata[blockSize];//must fix it size and don't use extern!!!
	sdata[tid] = ans;
	__syncthreads();
	if constexpr (blockSize == 1024)
	{
		if (tid < 512)
			sdata[tid] += sdata[tid + 512];
		__syncthreads();
	}
	if constexpr (blockSize >= 512)
	{
		if (tid < 256)
			sdata[tid] += sdata[tid + 256];
		__syncthreads();
	}
	if constexpr (blockSize >= 256)
	{
		if (tid < 128)
			sdata[tid] += sdata[tid + 128];
		__syncthreads();
	}
	if constexpr (blockSize >= 128)
	{
		if (tid < 64)
			sdata[tid] += sdata[tid + 64];
		__syncthreads();
	}
	/*if (tid < 32)
	{
		if constexpr (blockSize >= 64)ans = sdata[tid] + sdata[tid + 32];
		else ans = sdata[tid];
		if constexpr (blockSize >= 32)ans += __shfl_down_sync(0xffff, ans, 16);
		if constexpr (blockSize >= 16)ans += __shfl_down_sync(0xff, ans, 8);
		if constexpr (blockSize >= 8)ans += __shfl_down_sync(0xf, ans, 4);
		if constexpr (blockSize >= 4)ans += __shfl_down_sync(0x3, ans, 2);
		if constexpr (blockSize >= 2)ans += __shfl_down_sync(0x1, ans, 1);
	}*/
	if (tid < 32)warpReduce<T, blockSize>(sdata, tid);
	if (tid == 0)b[blockIdx.x] = sdata[0];
}
template<class T, unsigned long long blockSize>__global__ void reduction1(T* a, T* b, unsigned int N)//N % (4 * blockSize) == 0
{
	unsigned int const tid(threadIdx.x);
	unsigned int i(blockIdx.x * blockSize * 4 + tid);
	unsigned int const gridSize(4 * blockSize * gridDim.x);
	constexpr unsigned int warpNum(blockSize / 32);
	__shared__ T sdata[warpNum];//must fix it size and don't use extern!!!
	T ans(0);
	while (i < N)
	{
		ans += a[i + 3 * blockSize] + a[i + 2 * blockSize] + a[i + blockSize] + a[i];
		i += gridSize;
	}
	ans += __shfl_down_sync(0xffff, ans, 16);
	ans += __shfl_down_sync(0xff, ans, 8);
	ans += __shfl_down_sync(0xf, ans, 4);
	ans += __shfl_down_sync(0x3, ans, 2);
	ans += __shfl_down_sync(0x1, ans, 1);
	if ((tid & 31) == 0)sdata[tid / 32] = ans;
	__syncthreads();
	if (tid < 32)
	{
		if constexpr (warpNum >= 32)ans += __shfl_down_sync(0xffff, ans, 16);
		if constexpr (warpNum >= 16)ans += __shfl_down_sync(0xff, ans, 8);
		if constexpr (warpNum >= 8)ans += __shfl_down_sync(0xf, ans, 4);
		if constexpr (warpNum >= 4)ans += __shfl_down_sync(0x3, ans, 2);
		if constexpr (warpNum >= 2)ans += __shfl_down_sync(0x1, ans, 1);
		if (tid == 0)b[blockIdx.x] = ans;
	}
}


int main()
{
	Timer timer;
	std::mt19937 mt(time(0));
	std::uniform_int_distribution<unsigned int> rd(0, 5);

	constexpr unsigned long long N(1024llu * 1024llu * 2048llu);
	constexpr unsigned int gridDim(1024);
	constexpr unsigned long long aSize(N * sizeof(unsigned int));
	constexpr unsigned int bSize(gridDim * sizeof(unsigned int));

	unsigned int* a((unsigned int*)::malloc(aSize));
	if (a == nullptr)
	{
		::printf("malloc failed!");
		return -1;
	}
	unsigned int* aDevice;
	unsigned int* bDevice;
	unsigned int* answerDevice;
	unsigned int stdAnswer(0), answer;
	hipMalloc(&aDevice, aSize);
	hipMalloc(&bDevice, bSize);
	hipMalloc(&answerDevice, sizeof(unsigned int));
	::printf("Init numbers:\n");
	for (unsigned long long c0(0); c0 < N; ++c0)a[c0] = 1;// rd(mt);
	timer.begin();
	for (unsigned long long c0(0); c0 < N; ++c0)stdAnswer += a[c0];
	timer.end();
	timer.print("CPU:");
	::printf("%u\n", stdAnswer);

	hipMemcpy(aDevice, a, aSize, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	timer.begin();
	reduction1<unsigned int, 1024> << <1024, 1024 >> > (aDevice, bDevice, N - 1);
	reduction<unsigned int, 512> << <1, 512 >> > (bDevice, answerDevice, gridDim);
	hipDeviceSynchronize();
	timer.end();
	timer.print("GPU:");
	hipMemcpy(&answer, answerDevice, sizeof(unsigned int), hipMemcpyDeviceToHost);
	::printf("%u\n", answer);

	hipFree(aDevice);
	hipFree(bDevice);
	hipFree(answerDevice);
	free(a);
}